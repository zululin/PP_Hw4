
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int INF = 1000000000;
int V = 20010;
void input(char *inFileName);
void output(char *outFileName);

void block_FW(int B);
int ceil(int a, int b);

int n, m;	// Number of vertices, edges
int* host_ptr = NULL;
size_t pitch;

// for device
int* device_ptr = NULL;
__global__ void gpu_phase1(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, size_t pitch);
__global__ void gpu_phase2(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, int pos, size_t pitch);
__global__ void gpu_phase3(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, size_t pitch);

int main(int argc, char* argv[])
{
	input(argv[1]);
	int B = atoi(argv[3]);

	// allocate memory for device
	hipMallocPitch(&device_ptr, &pitch, V*sizeof(int), V);
	hipMemcpy2D(device_ptr, pitch, host_ptr, V*sizeof(int), V*sizeof(int), V, hipMemcpyHostToDevice);

	block_FW(B);

	hipMemcpy2D(host_ptr, V*sizeof(int), device_ptr, pitch, V*sizeof(int), V, hipMemcpyDeviceToHost);
	hipFree(device_ptr);

	output(argv[2]);
	free(host_ptr);

	return 0;
}

void input(char *inFileName)
{
	FILE *infile = fopen(inFileName, "r");
	fscanf(infile, "%d %d", &n, &m);

	// Malloc host memory
	V = n + 10;
	host_ptr = (int*)malloc((size_t)V * V * sizeof(int));

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			if (i == j)	host_ptr[i*V+j] = 0;
			else		host_ptr[i*V+j] = INF;
		}
	}

	while (--m >= 0) {
		int a, b, v;
		fscanf(infile, "%d %d %d", &a, &b, &v);
		host_ptr[a*V+b] = v;
	}
    fclose(infile);
}

void output(char *outFileName)
{
	FILE *outfile = fopen(outFileName, "w");
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
            if (host_ptr[i*V+j] >= INF)
                host_ptr[i*V+j] = INF;
		}
		fwrite(&host_ptr[i*V], sizeof(int), n, outfile);
	}
    fclose(outfile);
}

int ceil(int a, int b)
{
	return (a + b -1)/b;
}

void block_FW(int B)
{
	int round = ceil(n, B);
	dim3 blocks = {1, 1};
	dim3 threads = {(unsigned int)B, (unsigned int)B};

	printf("B: %d, Round: %d\n", B, round);

	for (unsigned int r = 0; r < round; ++r) {
		if (r % 10 == 0)
        	printf("%d %d\n", r, round);
		/* Phase 1*/
		blocks = {1, 1};
		gpu_phase1<<<blocks, threads, B*B*1*sizeof(int)>>>(device_ptr, B,	r,	r,	r, n, pitch/sizeof(int));

		/* Phase 2*/
		if (r > 0) {
			// left
			blocks = {1, r};
			gpu_phase2<<<blocks, threads, B*B*2*sizeof(int)>>>(device_ptr, B, r,     r,     0, n, 1, pitch/sizeof(int));

			// up
			blocks = {r, 1};
			gpu_phase2<<<blocks, threads, B*B*2*sizeof(int)>>>(device_ptr, B, r,     0,     r, n, 0, pitch/sizeof(int));
		}
		if (r < round - 1) {
			// right
			blocks = {1, round - r -1};
			gpu_phase2<<<blocks, threads, B*B*2*sizeof(int)>>>(device_ptr, B, r,     r,  r +1, n, 1, pitch/sizeof(int));

			// down
			blocks = {round - r -1, 1};
			gpu_phase2<<<blocks, threads, B*B*2*sizeof(int)>>>(device_ptr, B, r,  r +1,     r, n, 0, pitch/sizeof(int));
		}

		/* Phase 3*/
		if (r == 0) {
			// down right
			blocks = {round - r -1, round - r -1};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,  r +1,  r +1, n, pitch/sizeof(int));

		}
		else if (r == round - 1) {
			// upper left
			blocks = {r, r};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,     0,     0, n, pitch/sizeof(int));
		}
		else {
			// down right
			blocks = {round - r -1, round - r -1};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,  r +1,  r +1, n, pitch/sizeof(int));
			// upper left
			blocks = {r, r};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,     0,     0, n, pitch/sizeof(int));
			// upper right
			blocks = {r, round -r -1};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,     0,  r +1, n, pitch/sizeof(int));
			// down left
			blocks = {round - r -1, r};
			gpu_phase3<<<blocks, threads, B*B*3*sizeof(int)>>>(device_ptr, B, r,  r +1,     0, n, pitch/sizeof(int));
		}
	}
}

extern __shared__ int shared_mem[];
__global__ void gpu_phase1(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, size_t pitch)
{
	int V = pitch;
	int tid = threadIdx.y * B + threadIdx.x;
	int i = (block_start_x + blockIdx.x) * B + threadIdx.x;
	int j = (block_start_y + blockIdx.y) * B + threadIdx.y;

	// need self block - (b_i, b_j)
	shared_mem[tid] = dist[j*V+i];
	__syncthreads();

	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		if (i < n && j < n) {
			//===== change new posision by: =====//
			// new_i = origin_i - B * B_i
			// new_j = origin_j - B * B_j
			//===================================//
			int k_new = k - B * Round;
			int i_new = i - B * Round;
			int j_new = j - B * Round;

			int tmp = shared_mem[k_new*B+i_new] + shared_mem[j_new*B+k_new];

			if (tmp < shared_mem[tid]) {
				shared_mem[tid] = tmp;
			}
		}
		__syncthreads();
	}
	dist[j*V+i] = shared_mem[tid];
}

extern __shared__ int shared_mem[];
__global__ void gpu_phase2(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, int pos, size_t pitch)
{
	int V = pitch;
	int tid = threadIdx.y * B + threadIdx.x;
	int b_i = block_start_x + blockIdx.x;
	int b_j = block_start_y + blockIdx.y;
	int i = b_i * B + threadIdx.x;
	int j = b_j * B + threadIdx.y;

	// need self block - (b_i, b_j) & pivot block - (Round, Round)
	shared_mem[tid+B*B] = dist[j*V+i];
	shared_mem[tid] = dist[(Round*B+threadIdx.y)*V+(Round*B+threadIdx.x)];
	__syncthreads();

	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		if (i < n && j < n) {
			// up, down
			if (pos == 1) {
				int k_new_1 = k - B * Round;
				int i_new_1 = i - B * Round;

				int k_new_2 = k - B * b_i;
				int j_new_2 = j - B * b_j;

				int tmp = shared_mem[k_new_1*B+i_new_1] + shared_mem[j_new_2*B+k_new_2+B*B];

				if (tmp < shared_mem[tid+B*B]) {
					shared_mem[tid+B*B] = tmp;
				}
			}
			// left, right
			else {
				int k_new_1 = k - B * Round;
				int j_new_1 = j - B * Round;

				int i_new_2 = i - B * b_i;
				int k_new_2 = k - B * b_j;

				int tmp = shared_mem[k_new_2*B+i_new_2+B*B] + shared_mem[j_new_1*B+k_new_1];

				if (tmp < shared_mem[tid+B*B]) {
					shared_mem[tid+B*B] = tmp;
				}
			}
		}
		__syncthreads();
	}
	dist[j*V+i] = shared_mem[tid+B*B];
}

__global__ void gpu_phase3(int* dist, int B, int Round, int block_start_x, int block_start_y, int n, size_t pitch)
{
	int V = pitch;
	int tid = threadIdx.y * B + threadIdx.x;
	int b_i = block_start_x + blockIdx.x;
	int b_j = block_start_y + blockIdx.y;
	int i = b_i * B + threadIdx.x;
	int j = b_j * B + threadIdx.y;

	// need self block - (b_i, b_j) & row / column block
	shared_mem[tid] = dist[j*V+i];
	shared_mem[tid+B*B] = dist[(b_j*B+threadIdx.y)*V+(Round*B+threadIdx.x)];			// left, right
	shared_mem[tid+B*B*2] = dist[(Round*B+threadIdx.y)*V+(b_i*B+threadIdx.x)];			// up  , down
	__syncthreads();

	for (int k = Round * B; k < (Round +1) * B && k < n; ++k) {
		if (i < n && j < n) {
			// left, right
			int i_new_1 = i - B * b_i;
			int k_new_1 = k - B * Round;

			// up, down
			int k_new_2 = k - B * Round;
			int j_new_2 = j - B * b_j;

			int tmp = shared_mem[k_new_1*B+i_new_1+B*B*2] + shared_mem[j_new_2*B+k_new_2+B*B];

			if (tmp < shared_mem[tid]) {
				shared_mem[tid] = tmp;
			}
		}
		__syncthreads();
	}
	dist[j*V+i] = shared_mem[tid];
}
